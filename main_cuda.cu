#include "hip/hip_runtime.h"
// #include <iostream>
// #include <vector>
// using namespace std;

// __global__ void cuda_mul(vector<vector<float>> *vec1, vector<vector<float>> *vec2,vector<vector<float>> *res) {
//     for (int i = 0; i < vec1.size(); i++)
//     {
//         for (int j = 0; j < vec1[i].size(); j++)
//         {
//             res[i][j] = vec1[i][j] * vec2[i][j];
//         }
        
//     }
// }

// void getVec(vector<vector<float>> &vec1, vector<vector<float>> &vec2,vector<vector<float>> &res, int x)
// {

//     for (int i = 0; i < x; i++)
//     {
//         for (int j = 0; j < x; j++)
//         {
//             vec1[i][j] = rand() % 100 + 1;
//         }
//     }
//     for (int i = 0; i < x; i++)
//     {
//         for (int j = 0; j < x; j++)
//         {
//             vec2[i][j] = rand() % 100 + 1;
//         }
//     }
//     for (int i = 0; i < x; i++)
//     {
//         for (int j = 0; j < x; j++)
//         {
//             res[i][j] = rand() % 100 + 1;
//         }
//     }
// };

// int main()
// {
//     int x = 0;

//     cout << "Enter size of 2D Array: ";
//     cin >> x;

//     // vector<float> init(x);

//     // vector<vector<float>> *matrix1(x,init);

//     // vector<vector<float>> *matrix2(x,init);

//     // vector<vector<float>> *results(x,init);

//     hipMallocManaged(&matrix1,x*sizeof(float));
//     hipMallocManaged(&matrix2,x*sizeof(float));
//     hipMallocManaged(&results,x*sizeof(float));
//     getVec(matrix1,matrix2,results,x);

//     cuda_mul<<<1,1>>>(matrix1,matrix2,results);

//     hipDeviceSynchronize();

//     hipFree(matrix1);
//     hipFree(matrix2);
//     hipFree(results);
//     return -1;
// };

#include <hip/hip_runtime.h>
#include <iostream>
#include <ctime>

// âûïîëíÿåòñÿ íà GPU
__global__
void matmulDevice(int* A, int* B, int* C, int N)
{
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;

	if (row < N && col < N) {
		int sum = 0;
		for (int i = 0; i < N; i++)
			sum += A[row * N + i] * B[i * N + col];
		C[row * N + col] = sum;
	}
}

void matmulHost(int* A, int* B, int* C, int N)
{

	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			int sum = 0;
			for (int k = 0; k < N; k++)
				sum += A[i * N + k] * B[k * N + j];
			C[i * N + j] = sum;
		}
	}
}

#define checkCudaErrors(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

using namespace std;

int main(void)
{
	int device_count = 0;
	    hipGetDeviceCount(&device_count);
	
		if (device_count == 0)
			cout << "Sorry! You dont have CudaDevice" << endl;
		else
			cout << "CudaDevice found! Device count: " << device_count << endl;
	
		int N = 2048;
		int block_size = 16;
		// Êîë-âî èòåðàöèé
		int nIter = 1;

		unsigned int count = N*N;
		unsigned int mem_size = sizeof(int) * count;
	
	
		int* A = (int*)malloc(mem_size);
		int* B = (int*)malloc(mem_size);
		int* h_C = (int*)malloc(mem_size);
		int* hCuda_C = (int*)malloc(mem_size);
	
		int* d_A, * d_B, * d_C;
	
		for (int i = 0; i < count; i++) {
			A[i] = rand() % 100 + 1;
			B[i] = rand() % 100 + 1;
		}
	
		unsigned int start_time = clock();

		for (int j = 0; j < nIter; j++) {
			matmulHost(A, B, h_C, N);
		}

		unsigned int elapsedTime = clock() - start_time;
		float msecPerMatrixMulCpu = elapsedTime / nIter;

		cout << "CPU time: " << msecPerMatrixMulCpu << endl;
	
		checkCudaErrors(hipMalloc((void**)& d_A, mem_size));
		checkCudaErrors(hipMalloc((void**)& d_B, mem_size));
		checkCudaErrors(hipMalloc((void**)& d_C, mem_size));
	
		// êîïèðóåì äàííûå íà äåâàéñ
		checkCudaErrors(hipMemcpy(d_A, A, mem_size,
			hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(d_B, B, mem_size,
			hipMemcpyHostToDevice));
	
		dim3 threadsPerBlock(block_size, block_size);
		dim3 blocksPerGrid(N / block_size, N / block_size);
		
		hipEvent_t start;
		hipEvent_t stop;
		checkCudaErrors(hipEventCreate(&start));
		checkCudaErrors(hipEventCreate(&stop));

		// Çàïèñûâàåì íà÷àëî ñîáûòèÿ
		checkCudaErrors(hipEventRecord(start, 0));

		for (int j = 0; j < nIter; j++) {
			matmulDevice << <blocksPerGrid, threadsPerBlock >> > (d_A, d_B, d_C, N);
		}

		// Çàïèñûâàåì êîíåö ñîáûòèÿ
		checkCudaErrors(hipEventRecord(stop, 0));

		// Æäåì êîíöà ñîáûòèÿ
		checkCudaErrors(hipEventSynchronize(stop));

		float msecTotal = 0.0f;
		checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

		float msecPerMatrixMul = msecTotal / nIter;
			   
		cout << "GPU time: " << msecPerMatrixMul << endl;

		hipDeviceSynchronize();
	
		// êîïèðóåì ðåçóëüòàò ñ äåâàéñà
		checkCudaErrors(hipMemcpy(hCuda_C, d_C, mem_size, hipMemcpyDeviceToHost));
		hipDeviceSynchronize();
	
		// free device memory
		hipFree(d_A);
	    hipFree(d_B);
	    hipFree(d_C);
	
		bool test = true;
	
		for (int i = 0; i < count; i++) {
			if (h_C[i] != hCuda_C[i])
				test = false;
		}

		if (test)
			cout << "PASS!" << endl;
		else 
			cout << "WASTED!" << endl;
	
    return 0;
}

